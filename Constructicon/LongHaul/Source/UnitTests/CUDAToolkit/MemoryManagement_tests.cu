#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

namespace GoogleUnitTests
{
namespace CUDAToolkit
{

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(MemoryManagementTests, cudaMallocManagedAllocates)
{
  constexpr int N {1 << 20};
  float* x {nullptr};

  // Allocate Unified Memory -- accessible from CPU.
  hipMallocManaged(&x, N * sizeof(float));

  hipFree(x);

  SUCCEED();
}

} // namespace CUDAToolkit
} // namespace GoogleUnitTests
