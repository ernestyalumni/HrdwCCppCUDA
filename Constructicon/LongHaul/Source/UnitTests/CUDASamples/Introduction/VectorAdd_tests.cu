#include "hip/hip_runtime.h"
#include "CUDASamples/Introduction/VectorAddition.cuh"

#include <cmath>
#include <cstddef>
#include <cstdlib> // RAND_MAX
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
// See https://en.cppreference.com/w/c/memory/malloc
#include <stdlib.h> // malloc, rand()

using CUDASamples::Introduction::vector_add;
using std::size_t;

namespace GoogleUnitTests
{
namespace CUDAIntroduction
{

// See https://github.com/NVIDIA/cuda-samples/blob/master/Samples/0_Introduction/vectorAdd/vectorAdd.cu

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(VectorAddTests, AllocatesOnHostAndGPU)
{
  constexpr size_t number_of_elements {50000};
  constexpr size_t size {number_of_elements * sizeof(float)};

  // Allocate the host input vectors.

  float *h_A {static_cast<float*>(malloc(size))};
  float *h_B {static_cast<float*>(malloc(size))};
  float *h_C {static_cast<float*>(malloc(size))};

  EXPECT_NE(h_A, nullptr);
  EXPECT_NE(h_B, nullptr);
  EXPECT_NE(h_C, nullptr);

  for (size_t i {0}; i < number_of_elements; ++i)
  {
    // https://en.cppreference.com/w/cpp/numeric/random/RAND_MAX
    // RAND_MAX is implementation defined.
    h_A[i] = rand() / static_cast<float>(RAND_MAX);
    h_B[i] = rand() / static_cast<float>(RAND_MAX);
  }

  // Allocate device input vectors.

  float* d_A {nullptr};

  // See https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html
  // __hist__device__cudaError_t hipMalloc(void** devPtr, size_t size)
  hipError_t err {hipMalloc(reinterpret_cast<void**>(&d_A), size)};
  EXPECT_EQ(err, hipSuccess);

  float* d_B {nullptr};
  err = hipMalloc(reinterpret_cast<void**>(&d_B), size);
  EXPECT_EQ(err, hipSuccess);

  float* d_C {nullptr};
  err = hipMalloc(reinterpret_cast<void**>(&d_C), size);
  EXPECT_EQ(err, hipSuccess);

  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  EXPECT_EQ(err, hipSuccess);

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  EXPECT_EQ(err, hipSuccess);

  // Launch the Vector Add CUDA Kernel.
  constexpr size_t threads_per_block {256};
  constexpr size_t blocks_per_grid {
    (number_of_elements + threads_per_block - 1) / threads_per_block};  

  EXPECT_EQ(blocks_per_grid, 196);

  vector_add<<<blocks_per_grid, threads_per_block>>>(
    d_A,
    d_B,
    d_C,
    number_of_elements);

  err = hipGetLastError();
  EXPECT_EQ(err, hipSuccess);

  // Copy the device result vector in device memory to the host result vector in
  // host memory.
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  EXPECT_EQ(err, hipSuccess);

  // Verify that the result vector is correct
  for (size_t i {0}; i < number_of_elements; ++i)
  {
    EXPECT_TRUE(std::fabs(h_A[i] + h_B[i] - h_C[i]) < 1e-5);
  }

  err = hipFree(d_A);
  EXPECT_EQ(err, hipSuccess);

  err = hipFree(d_B);
  EXPECT_EQ(err, hipSuccess);

  err = hipFree(d_C);
  EXPECT_EQ(err, hipSuccess);

  // Free host memory.
  free(h_A);
  free(h_B);
  free(h_C);

  SUCCEED();
}

} // namespace CUDAIntroduction
} // namespace GoogleUnitTests
