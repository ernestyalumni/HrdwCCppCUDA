#include <cstddef>
#include <cstdlib> // RAND_MAX
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
// See https://en.cppreference.com/w/c/memory/malloc
#include <stdlib.h> // malloc, rand()

namespace GoogleUnitTests
{
namespace CUDAIntroduction
{

// See https://github.com/NVIDIA/cuda-samples/blob/master/Samples/0_Introduction/vectorAdd/vectorAdd.cu

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(VectorAddTests, AllocatesOnHostAndGPU)
{
  constexpr std::size_t number_of_elements {50000};
  constexpr std::size_t size {number_of_elements * sizeof(float)};

  // Allocate the host input vectors.

  float *h_A {static_cast<float*>(malloc(size))};
  float *h_B {static_cast<float*>(malloc(size))};
  float *h_C {static_cast<float*>(malloc(size))};

  EXPECT_NE(h_A, nullptr);
  EXPECT_NE(h_B, nullptr);
  EXPECT_NE(h_C, nullptr);

  for (std::size_t i {0}; i < number_of_elements; ++i)
  {
    // https://en.cppreference.com/w/cpp/numeric/random/RAND_MAX
    // RAND_MAX is implementation defined.
    h_A[i] = rand() / static_cast<float>(RAND_MAX);
    h_B[i] = rand() / static_cast<float>(RAND_MAX);
  }

  // Allocate device input vectors.

  float* d_A {nullptr};

  // See https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html
  // __hist__device__cudaError_t hipMalloc(void** devPtr, size_t size)
  hipError_t err {hipMalloc(reinterpret_cast<void**>(&d_A), size)};
  EXPECT_EQ(err, hipSuccess);

  float* d_B {nullptr};
  err = hipMalloc(reinterpret_cast<void**>(&d_B), size);
  EXPECT_EQ(err, hipSuccess);

  float* d_C {nullptr};
  err = hipMalloc(reinterpret_cast<void**>(&d_C), size);
  EXPECT_EQ(err, hipSuccess);

  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  EXPECT_EQ(err, hipSuccess);

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  EXPECT_EQ(err, hipSuccess);



  err = hipFree(d_A);
  EXPECT_EQ(err, hipSuccess);

  err = hipFree(d_B);
  EXPECT_EQ(err, hipSuccess);

  // Free host memory.
  free(h_A);
  free(h_B);
  free(h_C);

  SUCCEED();
}

} // namespace CUDAIntroduction
} // namespace GoogleUnitTests
